#include "CudaCheckError.h"
#include "CudaGraph.h"
#include <iostream>
#include <algorithm>

CudaLinkGraph::CudaLinkGraph(LinkGraph& memoryGraph)
{
    nodeNum = memoryGraph.getNodeNum();
    edgeNum = memoryGraph.getEdgeNum();

    // startEdgeIndices相当于原始va数组
    std::vector<nodeId_t> h_startEdgeIndices = memoryGraph.va;
    std::vector<nodeId_t> h_endEdgeIndices(nodeNum);

    for (nodeId_t i = 0; i < nodeNum; i++) {
        // 没有边的特殊处理
        if (h_startEdgeIndices[i] == NO_EDGE) {
            h_endEdgeIndices[i] = NO_EDGE;
            continue;
        }
        // 逐个去找终点
        nodeId_t curEdgeIndex = h_startEdgeIndices[i];
        nodeId_t nextNodeIndex = i + 1;
        while (nextNodeIndex < nodeNum && h_startEdgeIndices[nextNodeIndex] == NO_EDGE) {
            nextNodeIndex++;
        }
        // 如果超过了最后一个节点
        if (nextNodeIndex == nodeNum) {
            h_endEdgeIndices[i] = edgeNum;
        } else {
            h_endEdgeIndices[i] = h_startEdgeIndices[nextNodeIndex];
        }
    }
    std::cout << "startIndices: ";
    std::for_each(h_startEdgeIndices.begin(), h_startEdgeIndices.end(), [](int i) { std::cout << i << ' '; });
    std::cout << std::endl;
    std::cout << "endIndices: ";
    std::for_each(h_endEdgeIndices.begin(), h_endEdgeIndices.end(), [](int i) { std::cout << i << ' '; });
    std::cout << std::endl;
    
    checkError(hipMalloc(&d_edgeIndicesStart, edgeNum * sizeof(nodeId_t)));
    checkError(hipMemcpy(d_edgeIndicesStart, h_startEdgeIndices.data(), nodeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
    checkError(hipMalloc(&d_edgeIndicesEnd, edgeNum * sizeof(nodeId_t)));
    checkError(hipMemcpy(d_edgeIndicesEnd, h_endEdgeIndices.data(), nodeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
    checkError(hipMalloc(&d_ea, edgeNum * sizeof(nodeId_t)));
    checkError(hipMemcpy(d_ea, memoryGraph.ea.data(), edgeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
    checkError(hipMalloc(&d_weights, edgeNum * sizeof(weight_t)));
    checkError(hipMemcpy(d_weights, memoryGraph.weights.data(), edgeNum * sizeof(nodeId_t), hipMemcpyHostToDevice));
}

CudaLinkGraph::~CudaLinkGraph()
{
    checkError(hipFree(d_edgeIndicesStart));
    checkError(hipFree(d_edgeIndicesEnd));
    checkError(hipFree(d_ea));
    checkError(hipFree(d_weights));
}
